#include "hip/hip_runtime.h"
#include "PatchMatchFilter.cuh"
PatchMatchFilter::PatchMatchFilter(cv::Mat img1, cv::Mat img2)
{
	imSrcOrigin = img1.clone();
	imRefOrigin = img2.clone();
}

PatchMatchFilter::~PatchMatchFilter()
{
}

void PatchMatchFilter::Initialization(){
	
	//for kernel
	kernelSize = 9;
	
	subRadius = 2 * kernelSize;

	//for superpixel
	spNumber = 300;
	spSize = 300;
	spNumOrSize = 1;

	//for cost
	channels = 3;

	//for iteration
	iteration = 40;

	//initialize
	bestCost = (float *)malloc(sizeof(float)*imSrcOrigin.cols*imSrcOrigin.rows);
	bestFlow = (int2 *)malloc(sizeof(int2)*imSrcOrigin.cols*imSrcOrigin.rows);
}


void PatchMatchFilter::GetSuperpixelsListFromSegment(const cv::Mat_<int> &segLabels, int numOfLabels, std::vector<std::vector<cv::Vec2i>> &spPixelsList)
{
	int iy, ix, height, width;
	height = segLabels.rows;
	width = segLabels.cols;

	spPixelsList.clear();
	spPixelsList.resize(numOfLabels);
	for (iy = 0; iy<numOfLabels; ++iy)
		spPixelsList[iy].clear();
	for (iy = 0; iy<height; ++iy)
	{
		for (ix = 0; ix<width; ++ix)
		{
			int tmpLabel = segLabels[iy][ix];
			spPixelsList[tmpLabel].push_back(cv::Vec2i(iy, ix));
		}
	}
}



void PatchMatchFilter::CreateAndOrganizeSuperpixels(){
	cv::Mat im_src = imSrcOrigin, im_ref = imRefOrigin;
	cv::Mat_<int> labelSrc;
	cv::Mat_<cv::Vec4i> subSrc;//sub-image
	cv::Mat_<cv::Vec4i> spSrc;//superpixel

	int numLabelSrc;

	clock_t start, end;
	start = clock();

	numLabelSrc = CreateSLICSegments(im_src, labelSrc, spNumber, spSize, spNumOrSize);
	GetSubImageRangeFromSegments(labelSrc, numLabelSrc, subRadius, subSrc, spSrc);

	subRangeSrc = subSrc.clone();
	spRangeSrc = spSrc.clone();

	numOfLabelsSrc = numLabelSrc;
	segLabelsSrc = labelSrc.clone();

	GetSuperpixelsListFromSegment(segLabelsSrc, numOfLabelsSrc, superpixelsListSrc);

	/*********************** draw superpixel ********************************/
	//char *WINDOW_SEGMENT_CONTOUR = "segement_contour";
	//char *WINDOW_SEGMENT_CONTOUR_RIGHT = "segement_contour_right";
	//cvNamedWindow(WINDOW_SEGMENT_CONTOUR, CV_WINDOW_AUTOSIZE);
	//cvNamedWindow(WINDOW_SEGMENT_CONTOUR_RIGHT, CV_WINDOW_AUTOSIZE);
	//	
	//cv::Mat_<cv::Vec3b> resImg;
	//DrawContoursAroundSegments(im_src, labelSrc, resImg);
	//cv::imshow(WINDOW_SEGMENT_CONTOUR, resImg);
	//cv::Mat_<cv::Vec3b> resImgRef;
	//DrawContoursAroundSegments(im_ref, labelRef, resImgRef);
	//cv::imshow(WINDOW_SEGMENT_CONTOUR_RIGHT, resImgRef);
	//cv::imwrite("sup_src.png", resImg);
	//cv::imwrite("sup_ref.png", resImgRef);

	//cvWaitKey();
	//cv::destroyAllWindows();
	/*************************************************************************/

	end = clock();
	std::cout << "Finished creating superpixels, time :" << (end - start) / CLOCKS_PER_SEC << "s.\n";
}

void PatchMatchFilter::RandomAssignRepresentativePixel(const std::vector<std::vector<cv::Vec2i>> &spPixelsList, int numOfLabels, cv::Mat_<cv::Vec2i> &rePixel){
	
	rePixel.create(numOfLabels, 1);
	cv::RNG rng;
	int iy;
	for (iy = 0; iy<numOfLabels; ++iy)
	{
		rePixel[iy][0] = spPixelsList[iy][rng.next() % spPixelsList[iy].size()];
	}
}

void PatchMatchFilter::InitiateBufferData(){
	subImageSrc.clear();
	subImageSrc.resize(numOfLabelsSrc);

	for (int iy = 0; iy<numOfLabelsSrc; ++iy)
	{
		int py, px;
		py = repPixelsSrc[iy][0][0];
		px = repPixelsSrc[iy][0][1];
		// extract sub-image from subrange
		int w = subRangeSrc[py][px][2] - subRangeSrc[py][px][0] + 1;
		int h = subRangeSrc[py][px][3] - subRangeSrc[py][px][1] + 1;
		int x = subRangeSrc[py][px][0];
		int y = subRangeSrc[py][px][1];

		subImageSrc[iy] = imSrcOrigin(cv::Rect(x, y, w, h)).clone();
	}
}

void PatchMatchFilter::BuildSuperpixelsPropagationGraph(const cv::Mat_<int> &refSegLabel, int numOfLabels, const cv::Mat_<cv::Vec3f> &refImg, GraphStructure &spGraph)
{
	spGraph.adjList.clear();
	spGraph.vertexNum = 0;
	// build superpixel connectivity graph
	spGraph.ReserveSpace(numOfLabels * 20);
	spGraph.SetVertexNum(numOfLabels);
	int iy, ix, height, width;
	height = refSegLabel.rows;
	width = refSegLabel.cols;
	for (iy = 0; iy<height; ++iy)
	{
		for (ix = 0; ix<width; ++ix)
		{
			int tmp1 = refSegLabel[iy][ix];
			if (iy > 0)
			{
				int tmp2 = refSegLabel[iy - 1][ix];
				if (tmp1 != tmp2)
				{
					spGraph.AddEdge(tmp1, tmp2);
					//spGraph.AddEdge(tmp2, tmp1);
				}
			}

			if (ix > 0)
			{
				int tmp2 = refSegLabel[iy][ix - 1];
				if (tmp1 != tmp2)
				{
					spGraph.AddEdge(tmp1, tmp2);
					//spGraph.AddEdge(tmp2, tmp1);
				}
			}
		}
	}
}

void PatchMatchFilter::RunPatchMatchFilter(){

	RandomAssignRepresentativePixel(superpixelsListSrc, numOfLabelsSrc, repPixelsSrc);
	
	InitiateBufferData();//initialize some buffer data as sub image

	int iy, ix;

	spFlowVisitedNumberSrc.resize(numOfLabelsSrc);
	for (iy = 0; iy<numOfLabelsSrc; ++iy) spFlowVisitedNumberSrc[iy] = 0;

	spFlowVisitedSrc.resize(numOfLabelsSrc);
	for (iy = 0; iy<numOfLabelsSrc; ++iy) spFlowVisitedSrc[iy].clear();

	std::cout << "Now initializing..." << std::endl;

	for (iy = 0; iy < imSrcOrigin.rows; iy++)
	{
		for (ix = 0; ix < imSrcOrigin.cols; ix++)
		{
			bestFlow[iy*imSrcOrigin.cols + ix] = make_int2(0,0);
			bestCost[iy*imSrcOrigin.cols + ix] = DOUBLE_MAX;

		}
	}

	BuildSuperpixelsPropagationGraph(segLabelsSrc, numOfLabelsSrc, imSrcOrigin, spGraphSrc);

	for (iy = 0; iy < numOfLabelsSrc; iy++)
	{
		int ry = repPixelsSrc[iy][0][0];
		int rx = repPixelsSrc[iy][0][1];
		
		ImproveFlow(ry, rx, std::vector<int2>(1, make_int2(0, 0)));
	}

	for (int iter = 0; iter < iteration; iter++)
	{
		RandomAssignRepresentativePixel(superpixelsListSrc, numOfLabelsSrc, repPixelsSrc);

		int ystart = 0, yend = numOfLabelsSrc, ychange = 1;
		if (iter % 2 == 1)
		{
			ystart = numOfLabelsSrc - 1; yend = -1; ychange = -1;
		}

		for (iy = ystart; iy < yend; iy+=ychange)
		{
			std::vector<int2> dListVec;
			dListVec.clear();

			int refY, refX;
			refY = repPixelsSrc[iy][0][0];
			refX = repPixelsSrc[iy][0][1];

			std::set<int>::iterator sIt;
			std::set<int> &sAdj = spGraphSrc.adjList[iy];

			/* Propagation from neighbor */
			for (sIt = sAdj.begin(); sIt != sAdj.end(); sIt++)
			{
				repPixelsSrc[*sIt][0] = superpixelsListSrc[*sIt][rand() % superpixelsListSrc[*sIt].size()];
				int ky, kx;
				ky = repPixelsSrc[*sIt][0][0];
				kx = repPixelsSrc[*sIt][0][1];

				int2 tmpFlow = bestFlow[ky*imSrcOrigin.cols+kx];

				dListVec.push_back(tmpFlow);
			}

			/* Random search: Improve current guess by searching in boxes of exponentially decreasing size around the current best guess. */
			const float randomRatio = 2.0;

			float mag = std::min<float>(imRefOrigin.cols, imRefOrigin.rows);
			int2 tmpFlow = bestFlow[refY*imSrcOrigin.cols + refX];

			for (; mag >= 1; mag/=randomRatio)
			{
				float deltaHorLabel = (float(rand()) / RAND_MAX - 0.5)*2.0*mag;
				float deltaVerLabel = (float(rand()) / RAND_MAX - 0.5)*2.0*mag;

				float tmpHorLabel = tmpFlow.x + deltaHorLabel;
				float tmpVerLabel = tmpFlow.y + deltaVerLabel;

				tmpHorLabel = floor(tmpHorLabel + 0.5);
				tmpVerLabel = floor(tmpVerLabel + 0.5);

				if (tmpHorLabel < 0 || tmpHorLabel > imRefOrigin.cols || tmpVerLabel < 0 || tmpVerLabel > imRefOrigin.rows) continue;

				dListVec.push_back(make_int2(tmpHorLabel, tmpVerLabel));
			}
			ImproveFlow(refY, refX, dListVec);
		}

	}
}

void PatchMatchFilter::ImproveFlow(int py, int px, std::vector<int2> flowList){
	int flowSize = flowList.size();
	int w = subRangeSrc[py][px][2] - subRangeSrc[py][px][0] + 1;
	int h = subRangeSrc[py][px][3] - subRangeSrc[py][px][1] + 1;
	int x = subRangeSrc[py][px][0];
	int y = subRangeSrc[py][px][1];

	int dy, dx, ry, rx, oy, ox, cy, cx, cf, cc, sy, sx;

	float costTmp, costTotal = 0;

	//raw cost
	cv::Mat_<float> rawCost;
	rawCost.create(h, w*flowSize);

	for (int cf = 0; cf < flowSize; cf++)
	{
		int2 fl = flowList[cf];
		dy = fl.y;
		dx = fl.x;
		cv::Mat_<float> localRc = rawCost(cv::Rect(cf*w, 0, w, h));

		for (oy = y, cy = 0; cy < h; oy++, cy++)
		{
			for (ox = x, cx = 0; cx < w; ox++, cx++)
			{
				costTotal = 0;
				ry = oy + dy;
				rx = ox + dx;

				(ry < 0) ? ry = 0 : NULL;
				(ry >= imRefOrigin.rows) ? ry = imRefOrigin.rows - 1 : NULL;
				(rx < 0) ? rx = 0 : NULL;
				(rx >= imRefOrigin.cols) ? rx = imRefOrigin.cols - 1 : NULL;

				for (cc = 0; cc < channels; cc++)
				{
#if USE_COLOR_FEATURE
					costTmp = imSrcOrigin[oy][ox][cc] - imRefOrigin[ry][rx][cc];
					costTotal += costTmp*costTmp;
#endif // USE_COLOR_FEATURE

				}
				localRc[cy][cx] = costTotal;
			}
		}
	}

	//filtered cost
	cv::Mat_<float> filteredCost(h, w*flowSize);
	rawCost.copyTo(filteredCost);

	int spw = spRangeSrc[py][px][2] - spRangeSrc[py][px][0] + 1;
	int sph = spRangeSrc[py][px][3] - spRangeSrc[py][px][1] + 1;
	int spx = spRangeSrc[py][px][0];
	int spy = spRangeSrc[py][px][1];

	for (cf = 0; cf < flowSize; cf++)
	{
		oy = spy;
		sy = spy - y;
		for (cy = 0; cy < sph; cy++, oy++, sy++)
		{
			ox = spx;
			sx = spx - x;
			for (cx = 0; cx < spw; cx++, ox++, sx++)
			{
				costTmp = filteredCost[sy][sx + cf*w];
				if (costTmp < bestCost[oy*imSrcOrigin.cols+ox])
				{
					bestCost[oy*imSrcOrigin.cols + ox] = costTmp;
					bestFlow[oy*imSrcOrigin.cols + ox].x = flowList[cf].x;
					bestFlow[oy*imSrcOrigin.cols + ox].y = flowList[cf].y;
					//std::cout << "Updating x : " << ox << ", y : " << oy << " with flow : (" << flowList[cf].x << ", " << flowList[cf].y << ") and cost : " << costTmp << std::endl;
				}
			}
		}
	}
	

}


void PatchMatchFilter::ReconstructFlow(){
	cv::Mat flow = imSrcOrigin.clone();
	float maxVerRange = std::max<float>(imSrcOrigin.rows, imRefOrigin.rows);
	float maxHorRange = std::max<float>(imSrcOrigin.cols, imRefOrigin.cols);

	for (int iy = 0; iy < imRefOrigin.rows; iy++)
	{
		for (int ix = 0; ix < imRefOrigin.cols; ix++)
		{
			flow.at<cv::Vec3b>(iy, ix)[0] = 0;
			flow.at<cv::Vec3b>(iy, ix)[1] = (uchar)(255 * (bestFlow[iy*imSrcOrigin.cols + ix].x + maxHorRange) / (2 * maxHorRange));
			flow.at<cv::Vec3b>(iy, ix)[2] = (uchar)(255 * (bestFlow[iy*imSrcOrigin.cols + ix].y + maxVerRange) / (2 * maxVerRange));

		}
	}

	cv::imwrite(root + "flow.png", flow);
}

void PatchMatchFilter::ReconstructSrc(){
	cv::Mat res = imSrcOrigin.clone();

	int2 fl;
	int ry, rx;
	for (int iy = 0; iy < imRefOrigin.rows; iy++)
	{
		for (int ix = 0; ix < imRefOrigin.cols; ix++)
		{
			fl = bestFlow[iy*imSrcOrigin.cols+ix];
			ry = iy + fl.y;
			rx = ix + fl.x;
			res.at<cv::Vec3b>(iy, ix)[0] = imRefOrigin[ry][rx][0];
			res.at<cv::Vec3b>(iy, ix)[1] = imRefOrigin[ry][rx][1];
			res.at<cv::Vec3b>(iy, ix)[2] = imRefOrigin[ry][rx][2];

		}
	}

	cv::imwrite(root + "result.png", res);
}